#include <iostream>
#include <stdio.h>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;

void MatrixRandBin(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if ((float)rand()/RAND_MAX > 0.5) {
                mat[i*cols+j] = 1.0f;
            }else {
                mat[i*cols+j] = -1.0f;
            }

        }
    }
}

void MatrixPrint(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << setw(2) << mat[i*cols+j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

void MatrixPrintD(int *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << setw(2) << mat[i*cols+j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}


float MatrixCompare(float *a, float *b, int rows, int cols) {
    float err = 0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            err += abs(a[i*cols+j]-b[i*cols+j]);  
        }
    }
    return err;
}

void MatrixMul_host(float *a, int a_rows, int a_cols, float *b, int b_rows, int b_cols, float *c) {
    for (int i = 0; i < a_rows; i++) {
        for (int j = 0; j < b_cols; j++) {
            float t = 0;
            for (int k = 0; k < b_rows; k++) {
                t += a[i*a_cols+k]*b[k*b_cols+j];
            }
            c[i*b_cols+j] = t;
        }
    }
}

//horizontal
__global__ void AMatrix2Bin(float *a, int *a_bin, int pitch_a, int Pitch_a_bin, int a_rows, int MaxBlocks, int BINSIZE) {
    int tix = threadIdx.x;
    int bix = blockIdx.x;
    int bdx = blockDim.x;
    int gdx = gridDim.x;


    int maxThreads = MaxBlocks*a_rows;
    for (int id = bix*bdx+tix; id < maxThreads; id += gdx*bdx) {
        int rid = id/MaxBlocks;
        int cid = id%MaxBlocks;

        int Integer = 0;
        int base = 1;
        for (int i = 0; i < BINSIZE; i++) {
            if (a[rid*pitch_a+(cid+1)*BINSIZE-1-i] == 1.f) {
                Integer += base;
            }
            base = base<<1;
        }

        a_bin[rid*Pitch_a_bin+cid] = Integer;
    }

}
//vetical
__global__ void BMatrix2Bin(float *b, int *b_bin, int pitch_b, int Pitch_b_bin, int b_cols, int MaxBlocks, int BINSIZE) {
    int tix = threadIdx.x;
    int bix = blockIdx.x;
    int bdx = blockDim.x;
    int gdx = gridDim.x;

    int maxThreads = MaxBlocks*b_cols;
    for (int id = bix*bdx+tix; id < maxThreads; id += gdx*bdx) {
        int cid = id/MaxBlocks;
        int rid = id%MaxBlocks;

        int Integer = 0;
        int base = 1;
        for (int i=0; i < BINSIZE; i++) {
            if (b[((rid+1)*BINSIZE-1-i)*pitch_b+cid] == 1.f) {
                Integer += base;
            }
            base = base<<1;
        }

        b_bin[rid*Pitch_b_bin+cid] = Integer;
    }

}

// __device__ unsigned char __popcount_tab_copy[256];//__constant__ is slower than __device__
// __device__ int popcount (int x) {
//   return __popcount_tab_copy[(x >>  0) & 0xff]  
//   + __popcount_tab_copy[(x >>  8) & 0xff]  
//   + __popcount_tab_copy[(x >> 16) & 0xff] 
//   + __popcount_tab_copy[(x >> 24) & 0xff];
// }

__global__ void MatrixMulXnor(int *a, int *b, float *result, unsigned char *__popcount_tab,
 	int pitch_a, int pitch_b, int pitch_result,
	int midBlocks, int BINSIZE, int RealMidSize) {

    int tiy = threadIdx.x;
    int tix = threadIdx.y;
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int gdx = gridDim.x;
    int gdy = gridDim.y;
    int RectSize = blockDim.x;
    int rest = BINSIZE*RectSize*midBlocks-RealMidSize;

    __shared__ unsigned char __popcount_tab_shared[256];
    __shared__ int a_rect_shared[8][8];
    __shared__ int b_rect_shared[8][8];

    for (int i = tix*RectSize+tiy; i < 256; i += RectSize*RectSize) {
         __popcount_tab_shared[i] = __popcount_tab[i];
    }
    __syncthreads();

    int sum = 0;
    for (int i = 0; i < midBlocks; i++) {
    	a_rect_shared[tix][tiy] = a[(bix*RectSize+tix)*pitch_a+i*RectSize+tiy];
    	b_rect_shared[tix][tiy] = b[(i*RectSize+tix)*pitch_b+biy*RectSize+tiy];
    	__syncthreads();

    	int bin = 0;
    	bin = a_rect_shared[tix][0]^b_rect_shared[0][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][1]^b_rect_shared[1][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][2]^b_rect_shared[2][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][3]^b_rect_shared[3][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][4]^b_rect_shared[4][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][5]^b_rect_shared[5][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][6]^b_rect_shared[6][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    bin = a_rect_shared[tix][7]^b_rect_shared[7][tiy];
	    sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
						  + __popcount_tab_shared[(bin >>  8) & 0xff]  
						  + __popcount_tab_shared[(bin >> 16) & 0xff] 
						  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][8]^b_rect_shared[8][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][9]^b_rect_shared[9][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][10]^b_rect_shared[10][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][11]^b_rect_shared[11][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][12]^b_rect_shared[12][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][13]^b_rect_shared[13][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][14]^b_rect_shared[14][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    // bin = a_rect_shared[tix][15]^b_rect_shared[15][tiy];
	    // sum += BINSIZE-2*(	__popcount_tab_shared[(bin >>  0) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >>  8) & 0xff]  
					// 	  + __popcount_tab_shared[(bin >> 16) & 0xff] 
					// 	  + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][16]^b_rect_shared[16][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][17]^b_rect_shared[17][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][18]^b_rect_shared[18][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][19]^b_rect_shared[19][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][20]^b_rect_shared[20][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][21]^b_rect_shared[21][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][22]^b_rect_shared[22][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][23]^b_rect_shared[23][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][24]^b_rect_shared[24][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][25]^b_rect_shared[25][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][26]^b_rect_shared[26][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][27]^b_rect_shared[27][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][28]^b_rect_shared[28][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][29]^b_rect_shared[29][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][30]^b_rect_shared[30][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

     //  bin = a_rect_shared[tix][31]^b_rect_shared[31][tiy];
     //  sum += BINSIZE-2*(  __popcount_tab_shared[(bin >>  0) & 0xff]  
     //                    + __popcount_tab_shared[(bin >>  8) & 0xff]  
     //                    + __popcount_tab_shared[(bin >> 16) & 0xff] 
     //                    + __popcount_tab_shared[(bin >> 24) & 0xff]);

	    __syncthreads();
    }
    result[(bix*RectSize+tix)*pitch_result+biy*RectSize+tiy] = sum-rest;



    // num=0;
    // int rest=(BINSIZE*a_cols-RealMidSize);
    // for(int i=bix;i<a_rows;i+=gdx){
    //     for(int j=tix;j<b_cols;j+=bdx){
    //         // printf("i=%d ; j=%d\n",i,j);
    //         int sum=0;
    //         for(int k=0;k<a_cols;k++){
    //             int bin=(a_shared[num*a_cols+k]^b[k*pitch_b+j]);
    //             int negnum=popcount(bin);
    //             int posnum=BINSIZE-negnum;
    //             //calculate ignores the rest of BINSIZE if the Matsize can't devided by BINSIZE ,it can cause err
    //             //(10/00)'(01/00) should be 0000 but it is 0011,so 1+1 is trash in the result.and it mislead a_rows*b_cols times. 
    //             sum+=(posnum-negnum);
    //         }
    //         result[i*pitch_result+j]=sum-rest;
    //     }
    //     num++;
    // }


}


void MatrixMul_device(float *a, float *b, int a_rows, int a_cols, int b_cols, float *result) {

    int BINSIZE = 32;//size of bin2int, 32 means 0000 0000 0000 0000 0000 0000 0000 0000
    int MaxBlocks = (a_cols-1)/BINSIZE+1;
    int Copysize = MaxBlocks*BINSIZE;
    
    float *a_copy;//a_rows * Copysize
    float *b_copy;//Copysize * b_cols
    size_t Pitch_a_copy, Pitch_b_copy;
    hipMallocPitch((void**)&a_copy, &Pitch_a_copy, sizeof(float)*Copysize, a_rows);
    hipMallocPitch((void**)&b_copy, &Pitch_b_copy, sizeof(float)*b_cols, Copysize);
    hipMemset(a_copy, 0, Pitch_a_copy*a_rows);
    hipMemset(b_copy, 0, Pitch_b_copy*Copysize);
    hipMemcpy2D(a_copy, Pitch_a_copy, a, sizeof(float)*a_cols, sizeof(float)*a_cols, a_rows, hipMemcpyDeviceToDevice);
    hipMemcpy2D(b_copy, Pitch_b_copy, b, sizeof(float)*b_cols, sizeof(float)*b_cols, a_cols, hipMemcpyDeviceToDevice);

//check oringin
    // float *a_host;
    // float *b_host;
    // a_host = (float*) malloc(sizeof(float) * Copysize * a_rows);
    // b_host = (float*) malloc(sizeof(float) * b_cols * Copysize);
    // cudaMemcpy2D(a_host,sizeof(float) *Copysize, a_copy,Pitch_a_copy,sizeof(float) *Copysize , a_rows,cudaMemcpyDeviceToHost);
    // cudaMemcpy2D(b_host,sizeof(float) *b_cols, b_copy,Pitch_b_copy,sizeof(float) *b_cols , Copysize,cudaMemcpyDeviceToHost);
    // MatrixPrint(a_host,a_rows,Copysize);
    // MatrixPrint(b_host,Copysize,b_cols);


    int RectBlockSize = 4;
    dim3 RectBlockNum_a_bin((a_rows-1)/RectBlockSize+1, (MaxBlocks-1)/RectBlockSize+1, 1);//with block multiply
    dim3 RectBlockNum_b_bin((MaxBlocks-1)/RectBlockSize+1, (b_cols-1)/RectBlockSize+1, 1);
    int *a_bin;
    int *b_bin;
    size_t Pitch_a_bin, Pitch_b_bin;
    hipMallocPitch((void**)&a_bin , &Pitch_a_bin , sizeof(int)*RectBlockSize*RectBlockNum_a_bin.y, RectBlockSize*RectBlockNum_a_bin.x);
    hipMallocPitch((void**)&b_bin , &Pitch_b_bin , sizeof(int)*RectBlockSize*RectBlockNum_b_bin.y, RectBlockSize*RectBlockNum_b_bin.x);
    hipMemset(a_bin, 0, Pitch_a_bin*RectBlockSize*RectBlockNum_a_bin.x);
    hipMemset(b_bin, 0, Pitch_b_bin*RectBlockSize*RectBlockNum_b_bin.x);
    dim3 BS_BIN(512,1,1);
    dim3 GS_BIN(6,1,1);
    AMatrix2Bin<<< GS_BIN, BS_BIN >>>(a_copy, a_bin, 
        Pitch_a_copy/sizeof(float), Pitch_a_bin/sizeof(int), a_rows, MaxBlocks, BINSIZE);
    BMatrix2Bin<<< GS_BIN, BS_BIN >>>(b_copy, b_bin, 
        Pitch_b_copy/sizeof(float), Pitch_b_bin/sizeof(int), b_cols, MaxBlocks, BINSIZE);
    hipFree(a_copy);
    hipFree(b_copy);
//check bin
    // int *a_host_bin;
    // int *b_host_bin;
    // a_host_bin = (int*) malloc(sizeof(int) *MaxBlocks * a_rows);
    // b_host_bin = (int*) malloc(sizeof(int) *b_cols * MaxBlocks);
    // cudaMemcpy2D(a_host_bin,sizeof(int) *MaxBlocks, a_bin,Pitch_a_bin,sizeof(int) *MaxBlocks , a_rows ,cudaMemcpyDeviceToHost);
    // cudaMemcpy2D(b_host_bin,sizeof(int) *b_cols, b_bin,Pitch_b_bin,sizeof(int) *b_cols , MaxBlocks ,cudaMemcpyDeviceToHost);
    // MatrixPrintD(a_host_bin,a_rows,MaxBlocks);
    // MatrixPrintD(b_host_bin,MaxBlocks,b_cols);


    float *result_bin;//a_rows * b_cols
    size_t Pitch_result_bin;
    hipMallocPitch((void**)&result_bin , &Pitch_result_bin , sizeof(float)*RectBlockSize*RectBlockNum_b_bin.y, RectBlockSize*RectBlockNum_a_bin.x);

    const unsigned char __popcount_tab[] = {
      0,1,1,2,1,2,2,3,1,2,2,3,2,3,3,4,1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      1,2,2,3,2,3,3,4,2,3,3,4,3,4,4,5,2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      2,3,3,4,3,4,4,5,3,4,4,5,4,5,5,6,3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,
      3,4,4,5,4,5,5,6,4,5,5,6,5,6,6,7,4,5,5,6,5,6,6,7,5,6,6,7,6,7,7,8,
    };
    unsigned char *__popcount_tab_copy;
    hipMalloc((void**)&__popcount_tab_copy, sizeof(__popcount_tab));
    hipMemcpy(__popcount_tab_copy, __popcount_tab, sizeof(__popcount_tab), hipMemcpyHostToDevice);

    hipEvent_t start_device, stop_device;
    float time_device;
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);
    hipEventRecord(start_device, 0);

    dim3 BS_MM(RectBlockSize, RectBlockSize, 1);
    dim3 GS_MM(RectBlockNum_a_bin.x, RectBlockNum_b_bin.y, 1);
    MatrixMulXnor<<< GS_MM, BS_MM >>>(a_bin, b_bin, result_bin, __popcount_tab_copy,
     	Pitch_a_bin/sizeof(int), Pitch_b_bin/sizeof(int), Pitch_result_bin/sizeof(float),
     	RectBlockNum_a_bin.y, BINSIZE, a_cols);

    hipEventRecord( stop_device, 0 );
    hipEventSynchronize( stop_device );
    hipEventElapsedTime( &time_device, start_device, stop_device );
    hipEventDestroy( start_device );
    hipEventDestroy( stop_device );
    cout<<"gputime="<<time_device<<"ms"<<endl;

    hipMemcpy2D(result,sizeof(float) *b_cols, result_bin,Pitch_result_bin,sizeof(float) *b_cols , a_rows ,hipMemcpyDeviceToDevice);

    hipFree(a_bin);
    hipFree(b_bin);
    hipFree(result_bin);
}

int main(){

//simulate pytorch param
    int x=1000;
    int n=1000;
    int y=1000;
    float *a_host;
    float *b_host;
    float *result_host;
    a_host = (float*) malloc(sizeof(float) * x * n);
    b_host = (float*) malloc(sizeof(float) * n * y);
    result_host = (float*) malloc(sizeof(float) * x * y);
    srand(0);
    MatrixRandBin(a_host,x,n);
    MatrixRandBin(b_host,n,y);
    // cout<<MatrixCopysize<<endl;

    float *a_copy;
    float *b_copy;
    float *result_device;
    hipMalloc((void**)&a_copy,sizeof(float) *x * n);
    hipMalloc((void**)&b_copy,sizeof(float) *n * y);
    hipMalloc((void**)&result_device,sizeof(float) *x * y);
    hipMemcpy(a_copy,a_host,sizeof(float) *x * n,hipMemcpyHostToDevice);
    hipMemcpy(b_copy,b_host,sizeof(float) *n * y,hipMemcpyHostToDevice);


    // MatrixPrint(a_host,Matrixsize,Matrixsize);
    // MatrixPrint(b_host,Matrixsize,Matrixsize);

//run in gpu warp in C code
    MatrixMul_device(a_copy,b_copy,x,n,y,result_device);

    hipMemcpy(result_host, result_device,sizeof(float) *x * y,hipMemcpyDeviceToHost);
    hipFree(a_copy);
    hipFree(b_copy);
    hipFree(result_device);
    // MatrixPrint(result_host,Matrixsize,Matrixsize);

// //run in cpu
//     float *result_cpu;
//     result_cpu = (float*) malloc(sizeof(float) * x * y);
//     clock_t start_host = clock();
//     MatrixMul_host(a_host,x,n,b_host,n,y,result_cpu);
//     cout<<"cputime="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
//     // MatrixPrint(result_cpu,Matrixsize,Matrixsize);


// //compare value of gpu and cpu
//     float err=MatrixCompare(result_cpu,result_host,x,y);
//     cout<<"err in gpu and cpu = "<<err<<endl;

    return 0;
}