#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
int get_GPU_Rate()
{
	hipDeviceProp_t deviceProp;
 	hipGetDeviceProperties(&deviceProp,0);
 	return deviceProp.clockRate;
}

__global__ void Xor(int a,int b,int *result_device,clock_t* time){
	clock_t start = clock();
	int c;
	*result_device+=a^b;

	*time = clock() - start;
	
}


int main(){
	int *result_device;
    hipMalloc((void**) &result_device, sizeof(int));
    clock_t* time;
    hipMalloc((void**) &time, sizeof(clock_t));

    hipEvent_t start_device, stop_device;
    float time_device;
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);
    hipEventRecord( start_device, 0 );

    Xor<<<1,1>>>(1,-1,result_device,time);

	hipEventRecord( stop_device, 0 );
    hipEventSynchronize( stop_device );
    hipEventElapsedTime( &time_device, start_device, stop_device );
    hipEventDestroy( start_device );
    hipEventDestroy( stop_device );
    cout<<"gputime="<<time_device<<"ms"<<endl;


	clock_t time_used;
    hipMemcpy(&time_used, time, sizeof(clock_t),hipMemcpyDeviceToHost);
    cout<<"time="<<time_used<<endl;

    int result_host;
    hipMemcpy(&result_host, result_device, sizeof(int),hipMemcpyDeviceToHost);
    hipFree(result_device);
    cout<<result_host<<endl;

    return 0;

}

